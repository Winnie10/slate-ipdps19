#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hiprtc.h>

//threads per block
#define TPB 256
#define SCHED_ERR

#define NVRTC_SAFE_CALL(x) \
        do { \
            hiprtcResult result = x; \
            if (result != HIPRTC_SUCCESS) { \
                std::cerr << "\nerror: " #x " failed with error " \
                << hiprtcGetErrorString(result) << '\n'; \
                exit(1); \
            } \
         } while(0)

#define CUDA_SAFE_CALL(x) \
        do { \
            hipError_t result = x; \
            if (result != hipSuccess) { \
                const char *msg; \
                hipDrvGetErrorName(result, &msg); \
                std::cerr << "\nerror: " #x " failed with error " \
                << msg << '\n'; \
                exit(1); \
            } \
        } while(0) 

#ifdef SCHED_ERR
#define CHECK_CUDA_ERROR()                                                    \
{                                                                             \
    hipError_t err = hipGetLastError();                                     \
    if (err != hipSuccess)                                                   \
    {                                                                         \
        printf("error=%d name=%s at "                                         \
               "ln: %d\n  ",err,hipGetErrorString(err),__LINE__);            \
        exit(-1);                                                        \
    }                                                                         \
}
#else
#define CHECK_CUDA_ERROR() 
#endif

void get_props(hipDeviceProp_t* props)
{
    int count;
    hipGetDeviceCount(&count);
    CHECK_CUDA_ERROR();
    if (!count)
    {
        fprintf(stderr, "No devices found. Bye!\n");
        exit(1);
    }
    hipGetDeviceProperties(props, 0);
    CHECK_CUDA_ERROR();
}

void compile_and_run(int blocks)
{
    size_t ptxSize; 
    hipFunction_t kernel;
    hipModule_t module;
    hipCtx_t ctx;
    hipDevice_t cuDevice; 
    hiprtcLinkState linkState;

    hipDeviceGet(&cuDevice, 0);
    hipCtxCreate(&ctx, 0, cuDevice);


    hiprtcProgram prog;
    const char* buffer;
    const char *opts[] = {"-arch=compute_35", "-rdc=true"};
    std::ifstream in("test_code.cu");
    std::string contents((std::istreambuf_iterator<char>(in)), 
                          std::istreambuf_iterator<char>());
    contents.insert(0, "#include \"queue.h\"\n");
    //const char* header_name[] = { "queue.h" };
    //const char* header[] = { "__device__ void inc_val();\n__device__ int get_val();\n" };
    buffer = contents.c_str();
    NVRTC_SAFE_CALL(hiprtcCreateProgram(&prog, // prog 
                       buffer, // buffer 
                       "test_code.cu", // name 
                       0,//1, // numHeaders 
                       0,//header, // headers 
                       NULL));//header_name); // includeNames
    hiprtcResult compileResult = hiprtcCompileProgram(prog, // prog 
                                                    2, // numOptions 
                                                    opts); // options
    size_t logSize; 
    hiprtcGetProgramLogSize(prog, &logSize); 
    char *log = new char[logSize]; 
    hiprtcGetProgramLog(prog, log); 
    std::cerr << log << '\n';
    delete[] log;
    if (compileResult != HIPRTC_SUCCESS) 
    {
        fprintf(stderr, "Compile failed. Bye!\n");
        exit(1); 
    }
    CUDA_SAFE_CALL(hiprtcLinkCreate(0, 0, 0, &linkState));
    CUDA_SAFE_CALL(hiprtcLinkAddFile(linkState, hipJitInputObject, "queue.o", 0, 0, 0));

    NVRTC_SAFE_CALL(hiprtcGetCodeSize(prog, &ptxSize));
    char *ptx = new char[ptxSize];
    NVRTC_SAFE_CALL(hiprtcGetCode(prog, ptx));
    NVRTC_SAFE_CALL(hiprtcDestroyProgram(&prog));

    CUDA_SAFE_CALL(hiprtcLinkAddData(linkState, hipJitInputPtx, ptx, ptxSize, NULL, 0, 0, 0));
    size_t cubinSize; 
    void *cubin; 
    CUDA_SAFE_CALL(hiprtcLinkComplete(linkState, &cubin, &cubinSize));
    CUDA_SAFE_CALL(hipModuleLoadData(&module, cubin));

    CUDA_SAFE_CALL(hipModuleGetFunction(&kernel, module, "test1"));
    
    int* devp;
    hipMalloc(&devp, sizeof(int));
    void* args[] = { &devp };
    printf("Blocks: %d\nTPB:%d\n", blocks, TPB);
    hipModuleLaunchKernel(kernel,
                   blocks, 1, 1, // grid dim 
                   TPB, 1, 1, // block dim 
                   0, NULL, // shared mem and stream 
                   args, 0); // arguments
    hipDeviceSynchronize();
    int out = 0;
    hipMemcpy(&out, devp, sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    printf("Out: %d\n", out);
}

int main()
{
    int blocks;
    hipDeviceProp_t props;

    get_props(&props);
    blocks = (props.multiProcessorCount * props.maxThreadsPerMultiProcessor)/TPB; 
    
    compile_and_run(blocks);


    return 0;
}
