#include <iostream>
#include <hip/hip_runtime.h>

__global__ void glob()
{
    return;
}



int main()
{
    float time;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);  
    hipEventRecord(start, 0);
    glob<<<13, 128>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&time, start, stop);
    std::cout << time << std::endl;
    return 0;
}
