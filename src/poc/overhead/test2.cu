#include <iostream>
#include <hip/hip_runtime.h>

__global__ void gloob()
{
    return;
}

__global__ void glob()
{
    if (blockIdx.x == 0 && threadIdx.x == 0)
    {
       gloob<<<13, 128>>>(); 
    }
    return;
}

int main()
{
    float time;
    hipEvent_t start;
    hipEvent_t stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);  
    hipEventRecord(start, 0);
    glob<<<13, 128>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);  
    hipEventElapsedTime(&time, start, stop);
    std::cout << time << std::endl;
    return 0;
}
